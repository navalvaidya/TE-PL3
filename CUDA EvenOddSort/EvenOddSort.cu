#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 10
using namespace std;

__global__ void OddEvenSort(int *in,int *out,int size){
		
	bool OddEven = true;
	__shared__ bool swappedodd;
	__shared__ bool swappedeven;
	swappedodd=true;
	swappedeven=true;
	int temp;
	

	while(true){	
		if(OddEven){

			__syncthreads();
			swappedodd=false;
	
			int idx = blockIdx.x * blockDim.x + threadIdx.x;
			if(idx < size/2){
				if(in[idx*2] > in[idx*2+1]){
					//printf("\nThe odd swapping is: ");
					printf("\n\nThread id: %d \nodd Swapping %d <-> %d \nindexes [%d] <-> [%d]",idx,in[idx*2],in[idx*2+1],idx*2,(idx*2+1));				
					temp = in[idx*2];
					in[idx*2] = in[idx*2+1];
					in[idx*2+1] = temp;
					swappedodd=true;
				}
			}
			__syncthreads();			
		}
		else{
			__syncthreads();
			swappedeven=false;
	
			int idx = blockIdx.x * blockDim.x + threadIdx.x;
			
			if(idx < size/2-1){
				if(in[idx*2+1] > in[idx*2+2]){
					//printf("\nThe Even swapping is: ");
					printf("\n\nThread id: %d \nEven Swapping %d <-> %d \nindexes [%d] <-> [%d]",idx,in[idx*2+1],in[idx*2+2],(idx*2+1),(idx*2+2));				
					temp = in[idx*2+1];
					in[idx*2+1] = in[idx*2+2];
					in[idx*2+2] = temp;
					swappedeven=true;
				}
			}
			__syncthreads();	

		}

	if(!(swappedodd || swappedeven))
	break;		
	OddEven = !OddEven;	
	}
	__syncthreads();
	int idx = threadIdx.x;
	if(idx<size)
		out[idx] = in[idx];	
}

int main(){

	int *d_a,*d_asorted; 
	int size = sizeof(int) * SIZE;

	int *a = new int[size];
	int *asorted =new int[size];
	
	hipMalloc(&d_a,size);
	hipMalloc(&d_asorted,size);

	printf("\nEnter %d numbers to be sorted: \n",SIZE);
	for(int i=0;i<SIZE;i++){
		scanf("%d",&a[i]);	
	}

	printf("\nUnsorted array is: \n");
	for(int i=0;i<SIZE;i++){
		printf(" %d ",a[i]);	
	}

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	OddEvenSort<<<1,SIZE>>>(d_a,d_asorted,SIZE);
	hipMemcpy(asorted,d_asorted,size,hipMemcpyDeviceToHost);

	printf("\nSorted array is: \n");	
	for(int i=0;i<SIZE;i++){
	printf(" %d ",asorted[i]);
	}
	
	printf("\n\n");

	free(a);
	free(asorted);

	hipFree(d_a);
	hipFree(d_asorted);
return 0;
}
