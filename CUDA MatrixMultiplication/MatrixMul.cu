#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define BLOCK_SIZE 2
#define GRID_SIZE 2
#define N GRID_SIZE*BLOCK_SIZE
using namespace std;

__global__ void MatrixMul(float *d_a,float *d_b,float *d_c,int n){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	float sum=0;

	for(int i=0;i<n;i++){
		sum += (d_a[row * n + i] * d_b[i * n + column]);
	}

	d_c[row*n+column] = sum;
	printf("\nBlock Id [%d][%d], Thread Id [%d][%d], Product : %.2f ",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,sum);	
	
}



int main(){

	float *d_a,*d_b,*d_c;
	int n = N * N * sizeof(float);

	float *a = new float[n];
	float *b = new float[n];
	float *c = new float[n];

	for(int j=0;j<N;j++){
		for(int i=0;i<N;i++){
			a[j*N+i] = 2*i+4*j+1;
			b[j*N+i] = 3*i+2*j+1;	 	
		}	
	}

	printf("\nMatrix 1 is:\n");
	for(int j=0;j<N;j++){
		for(int i=0;i<N;i++){
			printf(" %.2f ",a[j*N+i]);	 	
		}
		printf("\n");	
	}

	printf("\nMatrix 2 is:\n");
	for(int j=0;j<N;j++){
		for(int i=0;i<N;i++){
			printf(" %.2f ",b[j*N+i]);	 	
		}
		printf("\n");	
	}

	hipMalloc(&d_a,n);
	hipMalloc(&d_b,n);
	hipMalloc(&d_c,n);

	hipMemcpy(d_a,a,n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n,hipMemcpyHostToDevice);

	dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(GRID_SIZE,GRID_SIZE);

	MatrixMul<<<grid,threadBlock>>>(d_a,d_b,d_c,N);

	hipMemcpy(c,d_c,n,hipMemcpyDeviceToHost);
	printf("\nMatrix Multiplication:\n");
	for(int j=0;j<N;j++){
		for(int i=0;i<N;i++){
			printf(" %.2f ",c[j*N+i]);	 	
		}
		printf("\n");	
	}	
	free(a);
	free(b);
	free(c);

	hipFree(a);
	hipFree(b);
	hipFree(c);


	return 0;
}


