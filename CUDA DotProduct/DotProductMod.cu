#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#define SIZE 10
using namespace std;

__global__ void DotProduct(int *d_a,int *d_b,int *d_c){

	*d_c=0;
	int sum=0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sum += d_a[idx]*d_b[idx];	
	printf("Thread ID : %d\n",idx);
	printf(":%d ",sum);
	atomicAdd(d_c,sum);

}

int main(){
	int *d_a,*d_b,*d_c;
	int n = SIZE*sizeof(int);

	int *a = new int[n];
	int *b = new int[n];
	int *c = new int[n];

	hipMalloc(&d_a,n);
	hipMalloc(&d_b,n);
	hipMalloc(&d_c,sizeof(int));

	for(int i=0;i<SIZE;i++){
		a[i]=i*2;	
		b[i]=(i+1)*2;
	}

	hipMemcpy(d_a,a,n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n,hipMemcpyHostToDevice);

	DotProduct <<<1,SIZE>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"\nDot Product is : "<<*c<<"\n";
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(a);
	free(b);
	free(c);
return 0;
}
