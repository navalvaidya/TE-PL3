#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#define SIZE 10
using namespace std;

__global__ void DotProduct(int *d_a,int *d_b,int *d_c)
{
	__shared__ int temp[SIZE];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	temp[threadIdx.x] = d_a[tid] * d_b[tid];
	printf("%d:",temp[tid]);
	printf("\nThread id is: %d",threadIdx.x);
	__syncthreads();

	if(tid==0){
		int sum=0;
		for(int i=0;i<SIZE;i++){
			sum += temp[i];
		}	
	*d_c = sum;
	printf("\nBlock id is: %d",blockIdx.x);
	printf("\nSum is:%d",*d_c);
	}

}


int main(){

	int *d_a,*d_b,*d_c;
	int n = sizeof(int) * SIZE;

	int *a = new int[n];
	int *b = new int[n];
	int *c = new int[n];

	for(int i=0;i<SIZE;i++){
		a[i]=i*2;
		b[i]=(i+1)*2;
	}

	hipMalloc(&d_a,n);
	hipMalloc(&d_b,n);
	hipMalloc(&d_c,sizeof(int));

	hipMemcpy(d_a,a,n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n,hipMemcpyHostToDevice);
		
	DotProduct<<<1,SIZE>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,sizeof(int),hipMemcpyDeviceToHost);
	printf("\nDot product is: %d\n",*c);

	free(a);
	free(b);
	free(c);


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
return 0;
}
